#include "hip/hip_runtime.h"
#include <cstddef>
#include <iostream>
#include <algorithm>
#pragma once

#include "ctc_helper.h"
#include "gpu_ctc_kernels.h"
#include "reduce.cuh"

template <typename ProbT>
class GpuCTC
{
public:
  GpuCTC(int alphabet_size,
    int minibatch,
    void* workspace,
    hipStream_t stream,
    int blank_label) :
    out_dim_(alphabet_size), minibatch_(minibatch),
    gpu_workspace_(workspace), stream_(stream),
    blank_label_(blank_label) {};

  // Noncopyable
  GpuCTC(const GpuCTC &) = delete;
  GpuCTC & operator=(const GpuCTC &) = delete;

  ctcStatus_t
    cost_and_grad(const ProbT* const activations,
      ProbT* grads,
      ProbT* costs,
      const int* const flat_labels,
      const int* const label_lengths,
      const int* const input_lengths);

  ctcStatus_t
    score_forward(const ProbT* const activations,
      ProbT* costs,
      const int* const flat_labels,
      const int* const label_lengths,
      const int* const input_lengths);

private:

  template<int NT, int VT>
  ctcStatus_t launch_alpha_beta_kernels(const ProbT* const probs,
    ProbT* grads,
    bool compute_alpha,
    bool compute_beta);

  ctcStatus_t
    launch_gpu_kernels(const ProbT* const probs,
      ProbT* grads,
      size_t config,
      bool launch_alpha,
      bool launch_beta);

  ctcStatus_t
    setup_gpu_metadata(const int* const flat_labels,
      const int* const label_lengths,
      const int* const input_lengths);

  ctcStatus_t
    create_metadata_and_choose_config(const int* const label_lengths,
      const int* const flat_labels,
      const int* const input_lengths,
      size_t & best_config);

  ctcStatus_t
    compute_probs(const ProbT* const activations);

  ctcStatus_t
    compute_cost_and_score(const ProbT* const activations,
      ProbT* grads,
      ProbT* costs,
      const int* const flat_labels,
      const int* const label_lengths,
      const int* const input_lengths,
      bool compute_alpha,
      bool compute_betas_and_grad);


  int out_dim_; // Number of characters plus blank
  int minibatch_;

  int S_;
  int T_;

  int activation_cols_; // Number of columns in activations

  hipStream_t stream_;
  int blank_label_;

  void* gpu_workspace_; // Buffer for all temporary GPU memory
  int* utt_length_; // T
  int* label_sizes_; // L
  int* repeats_; // repeats_
  int* label_offsets_;
  int* labels_without_blanks_;
  int* labels_with_blanks_;
  ProbT* alphas_;
  ProbT* nll_forward_;
  ProbT* nll_backward_;
  ProbT* denoms_; // Temporary storage for denoms for softmax
  ProbT* probs_; // Temporary storage for probabilities (softmax output)
};

template<typename ProbT>
ctcStatus_t
GpuCTC<ProbT>::setup_gpu_metadata(const int* const flat_labels,
  const int* const label_lengths,
  const int* const input_lengths)
{
  size_t gpu_bytes_used = 0;
  nll_forward_ =
    reinterpret_cast<ProbT*>(static_cast<char*>(gpu_workspace_) +
      gpu_bytes_used);
  gpu_bytes_used += minibatch_ * sizeof(ProbT);
  nll_backward_ =
    reinterpret_cast<ProbT*>(static_cast<char*>(gpu_workspace_) +
      gpu_bytes_used);
  gpu_bytes_used += minibatch_ * sizeof(ProbT);
  repeats_ =
    reinterpret_cast<int*>(static_cast<char*>(gpu_workspace_) +
      gpu_bytes_used);
  gpu_bytes_used += minibatch_ * sizeof(int);
  label_offsets_ =
    reinterpret_cast<int*>(static_cast<char*>(gpu_workspace_) +
      gpu_bytes_used);
  gpu_bytes_used += minibatch_ * sizeof(int);
  // This is the max of all S and T for all valid examples in the minibatch.
  // A valid example is one for which L + repeats <= T
  S_ = 0;
  T_ = 0;
  // This is the max of all timesteps, valid or not. Needed to compute offsets
  int Tmax = 0;
  // This is the max of all labels, valid or not. Needed to compute offsets
  int Lmax = 0;
  int total_label_length = 0;
  constexpr int cpu_buffer_size = 64;
  int repeats[cpu_buffer_size];
  int label_offsets[cpu_buffer_size];
  const int num_passes = ctc_helper::div_up(minibatch_, cpu_buffer_size);
  hipError_t cuda_status;
  for (int pass = 0; pass < num_passes; ++pass) {
    const int start_idx = pass * cpu_buffer_size;
    const int end_idx = std::min(minibatch_, (pass + 1) * cpu_buffer_size);
    for (int j = start_idx; j < end_idx; ++j) {
      const int L = label_lengths[j];
      const int local_T = input_lengths[j];
      const int* label_ptr = &(flat_labels[total_label_length]);
      label_offsets[j % cpu_buffer_size] = total_label_length;
      total_label_length += L;
      int repeat_counter = 0;
      for (int i = 1; i < L; ++i) {
        repeat_counter += (label_ptr[i] == label_ptr[i - 1]);
      }
      repeats[j % cpu_buffer_size] = repeat_counter;
      const bool valid_label = ((L + repeat_counter) <= local_T);
      // Only update S and T if label is valid
      S_ = (valid_label) ? std::max(S_, L) : S_;
      T_ = (valid_label) ? std::max(T_, local_T) : T_;
      Tmax = std::max(Tmax, local_T);
      Lmax = std::max(Lmax, L);
    }
    cuda_status = hipMemcpyAsync(&(repeats_[start_idx]), repeats,
      (end_idx - start_idx) * sizeof(int),
      hipMemcpyHostToDevice, stream_);
    if (cuda_status != hipSuccess) {
      return CTC_STATUS_MEMOPS_FAILED;
    }
    cuda_status = hipMemcpyAsync(&(label_offsets_[start_idx]), label_offsets,
      (end_idx - start_idx) * sizeof(int),
      hipMemcpyHostToDevice, stream_);
    if (cuda_status != hipSuccess) {
      return CTC_STATUS_MEMOPS_FAILED;
    }
  }
  S_ = 2 * S_ + 1;
  const int Smax = 2 * Lmax + 1;
  activation_cols_ = minibatch_ * Tmax;
  // Allocate memory for T
  utt_length_ =
    reinterpret_cast<int*>(static_cast<char*>(gpu_workspace_) +
      gpu_bytes_used);
  gpu_bytes_used += minibatch_ * sizeof(int);
  cuda_status = hipMemcpyAsync(utt_length_, input_lengths,
    minibatch_ * sizeof(int),
    hipMemcpyHostToDevice, stream_);
  if (cuda_status != hipSuccess) {
    return CTC_STATUS_MEMOPS_FAILED;
  }
  label_sizes_ =
    reinterpret_cast<int*>(static_cast<char*>(gpu_workspace_) +
      gpu_bytes_used);
  gpu_bytes_used += minibatch_ * sizeof(int);
  cuda_status = hipMemcpyAsync(label_sizes_, label_lengths,
    minibatch_ * sizeof(int),
    hipMemcpyHostToDevice, stream_);
  if (cuda_status != hipSuccess) {
    return CTC_STATUS_MEMOPS_FAILED;
  }
  labels_without_blanks_ =
    reinterpret_cast<int*>(static_cast<char*>(gpu_workspace_) +
      gpu_bytes_used);
  gpu_bytes_used += Lmax * minibatch_ * sizeof(int);
  cuda_status = hipMemcpyAsync(labels_without_blanks_, flat_labels,
    total_label_length * sizeof(int),
    hipMemcpyHostToDevice, stream_);
  if (cuda_status != hipSuccess) {
    return CTC_STATUS_MEMOPS_FAILED;
  }
  labels_with_blanks_ =
    reinterpret_cast<int*>(static_cast<char*>(gpu_workspace_) +
      gpu_bytes_used);
  gpu_bytes_used += Smax * minibatch_ * sizeof(int);
  alphas_ =
    reinterpret_cast<ProbT*>(static_cast<char*>(gpu_workspace_) +
      gpu_bytes_used);
  gpu_bytes_used += (S_ * T_) * minibatch_ * sizeof(ProbT);
  denoms_ =
    reinterpret_cast<ProbT*>(static_cast<char*>(gpu_workspace_) +
      gpu_bytes_used);
  gpu_bytes_used += activation_cols_ * sizeof(ProbT);
  probs_ =
    reinterpret_cast<ProbT*>(static_cast<char*>(gpu_workspace_) +
      gpu_bytes_used);
  gpu_bytes_used += out_dim_ * activation_cols_ * sizeof(ProbT);
  return CTC_STATUS_SUCCESS;
}

template<typename ProbT>
template<int NT, int VT>
ctcStatus_t GpuCTC<ProbT>::launch_alpha_beta_kernels(const ProbT* const probs,
  ProbT* grads,
  bool compute_alpha,
  bool compute_beta)
{
  // One thread block per utterance
  const int grid_size = minibatch_;
  // The data is laid out so that the next timestep is minibatch entries
  // away
  const int stride = minibatch_;
  if (compute_alpha)
    compute_alpha_kernel<ProbT, NT, VT> << < grid_size, NT, 0, stream_ >> >
    (probs, label_sizes_, utt_length_,
      repeats_, labels_without_blanks_, label_offsets_,
      labels_with_blanks_, alphas_, nll_forward_,
      stride, out_dim_, S_, T_, blank_label_);
  if (compute_beta) {
    compute_betas_and_grad_kernel<ProbT, NT, VT> << < grid_size, NT, 0, stream_ >> >
      (probs, label_sizes_, utt_length_, repeats_,
        labels_with_blanks_, alphas_, nll_forward_, nll_backward_,
        grads, stride, out_dim_, S_, T_, blank_label_);
    hipStreamSynchronize(stream_);
  }
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
    return CTC_STATUS_EXECUTION_FAILED;
  return CTC_STATUS_SUCCESS;
}

template<typename ProbT>
ctcStatus_t
GpuCTC<ProbT>::create_metadata_and_choose_config(const int* const flat_labels,
  const int* const label_lengths,
  const int* const input_lengths,
  size_t & best_config)
{
  // Setup the metadata for GPU
  ctcStatus_t status = setup_gpu_metadata(flat_labels, label_lengths, input_lengths);
  if (status != CTC_STATUS_SUCCESS) {
    return status;
  }
  constexpr int num_configs = 12;
  int config_NT[num_configs] =
  { 32, 64, 128, 64, 128, 32, 64, 128, 64, 128, 128, 128 };
  int config_VT[num_configs] =
  { 1,  1,   1,  3,   2,  9,  6,   4,  9,   6,   9,  10 };
  best_config = 0;
  for (int i = 0; i < num_configs; ++i) {
    if ((config_NT[i] * config_VT[i]) >= S_) {
      break;
    }
    else {
      best_config++;
    }
  }
  if (best_config >= num_configs) {
    return CTC_STATUS_UNKNOWN_ERROR;
  }
  return CTC_STATUS_SUCCESS;
}

template<typename ProbT>
ctcStatus_t
GpuCTC<ProbT>::launch_gpu_kernels(const ProbT* const probs,
  ProbT* grads,
  size_t config,
  bool l_a,
  bool l_b)
{
  switch (config) {
  case 0:
  {return launch_alpha_beta_kernels<32, 1>(probs, grads, l_a, l_b); }
  case 1:
  {return launch_alpha_beta_kernels<64, 1>(probs, grads, l_a, l_b); }
  case 2:
  {return launch_alpha_beta_kernels<128, 1>(probs, grads, l_a, l_b); }
  case 3:
  {return launch_alpha_beta_kernels<64, 3>(probs, grads, l_a, l_b); }
  case 4:
  {return launch_alpha_beta_kernels<128, 2>(probs, grads, l_a, l_b); }
  case 5:
  {return launch_alpha_beta_kernels<32, 9>(probs, grads, l_a, l_b); }
  case 6:
  {return launch_alpha_beta_kernels<64, 6>(probs, grads, l_a, l_b); }
  case 7:
  {return launch_alpha_beta_kernels<128, 4>(probs, grads, l_a, l_b); }
  case 8:
  {return launch_alpha_beta_kernels<64, 9>(probs, grads, l_a, l_b); }
  case 9:
  {return launch_alpha_beta_kernels<128, 6>(probs, grads, l_a, l_b); }
  case 10:
  {return launch_alpha_beta_kernels<128, 9>(probs, grads, l_a, l_b); }
  case 11:
  {return launch_alpha_beta_kernels<128, 10>(probs, grads, l_a, l_b); }
  }
  return CTC_STATUS_EXECUTION_FAILED;
}

template<typename ProbT>
ctcStatus_t
GpuCTC<ProbT>::compute_probs(const ProbT* const activations)
{
  hipError_t cuda_status;
  cuda_status =
    hipMemcpyAsync(probs_, activations,
      activation_cols_ * out_dim_ * sizeof(ProbT),
      hipMemcpyDeviceToDevice, stream_);
  if (cuda_status != hipSuccess) {
    return CTC_STATUS_MEMOPS_FAILED;
  }
  // Numerically stable SM
  ctcStatus_t ctc_status =
    reduce_max(probs_, denoms_, out_dim_,
      activation_cols_, 1, stream_);
  if (ctc_status != CTC_STATUS_SUCCESS) {
    return ctc_status;
  }
  // Kernel launch to subtract maximum
  const int NT = 128;
  const int VT = 1;
  const int NV = NT * VT;
  const int num_elements = out_dim_ * activation_cols_;
  const int grid_size = ctc_helper::div_up(num_elements, NV);
  prepare_stable_SM_kernel<ProbT, VT> << < grid_size, NT, 0, stream_ >> >
    (ctc_helper::identity<ProbT>(), probs_,
      denoms_, out_dim_, num_elements);
  // Reduce along columns to calculate denominator
  ctc_status =
    reduce_exp(probs_, denoms_, out_dim_,
      activation_cols_, 1, stream_);
  if (ctc_status != CTC_STATUS_SUCCESS)
    return ctc_status;
  // Kernel launch to calculate probabilities
  compute_probs_kernel<ProbT, VT> << < grid_size, NT, 0, stream_ >> >
    (ctc_helper::exponential<ProbT>(), probs_,
      denoms_, out_dim_, num_elements);
  return CTC_STATUS_SUCCESS;
}

template<typename ProbT>
ctcStatus_t
GpuCTC<ProbT>::compute_cost_and_score(const ProbT* const activations,
  ProbT* grads,
  ProbT* costs,
  const int* const flat_labels,
  const int* const label_lengths,
  const int* const input_lengths,
  bool compute_alpha,
  bool compute_betas_and_grad)
{
  size_t best_config;
  ctcStatus_t status = create_metadata_and_choose_config(flat_labels,
    label_lengths,
    input_lengths,
    best_config);
  if (status != CTC_STATUS_SUCCESS) {
    return status;
  }
  status = compute_probs(activations);
  if (status != CTC_STATUS_SUCCESS) {
    return status;
  }
  launch_gpu_kernels(probs_, grads, best_config,
    compute_alpha, compute_betas_and_grad);
  hipError_t cuda_status_mem, cuda_status_sync;
  cuda_status_mem = hipMemcpyAsync(costs, nll_forward_,
    sizeof(ProbT) * minibatch_,
    hipMemcpyDeviceToHost, stream_);
  cuda_status_sync = hipStreamSynchronize(stream_);
  if (cuda_status_mem != hipSuccess || cuda_status_sync != hipSuccess) {
    return CTC_STATUS_MEMOPS_FAILED;
  }
  return CTC_STATUS_SUCCESS;
}

template<typename ProbT>
ctcStatus_t
GpuCTC<ProbT>::cost_and_grad(const ProbT* const activations,
  ProbT* grads,
  ProbT* costs,
  const int* const flat_labels,
  const int* const label_lengths,
  const int* const input_lengths)
{
  if (activations == nullptr ||
    grads == nullptr ||
    costs == nullptr ||
    flat_labels == nullptr ||
    label_lengths == nullptr ||
    input_lengths == nullptr
    ) {
    return CTC_STATUS_INVALID_VALUE;
  }
  return compute_cost_and_score(activations, grads, costs, flat_labels,
    label_lengths, input_lengths, true, true);
}

template<typename ProbT>
ctcStatus_t
GpuCTC<ProbT>::score_forward(const ProbT* const activations,
  ProbT* costs,
  const int* const flat_labels,
  const int* const label_lengths,
  const int* const input_lengths)
{
  if (activations == nullptr ||
    costs == nullptr ||
    flat_labels == nullptr ||
    label_lengths == nullptr ||
    input_lengths == nullptr
    ) {
    return CTC_STATUS_INVALID_VALUE;
  }
  return compute_cost_and_score(activations, nullptr, costs, flat_labels,
    label_lengths, input_lengths, true, false);
}



ctcStatus_t FUN(compute_ctc_loss)(const Dtype* const activations,
  Dtype* gradients,
  const int* const flat_labels,
  const int* const label_lengths,
  const int* const input_lengths,
  int alphabet_size,
  int minibatch,
  Dtype *costs,
  void *workspace,
  ctcOptions options) {

  if (activations == nullptr ||
    flat_labels == nullptr ||
    label_lengths == nullptr ||
    input_lengths == nullptr ||
    costs == nullptr ||
    workspace == nullptr ||
    alphabet_size <= 0 ||
    minibatch <= 0)
    return CTC_STATUS_INVALID_VALUE;

  GpuCTC<Dtype> ctc(alphabet_size, minibatch, workspace, options.stream,
    options.blank_label);

  if (gradients != NULL)
    return ctc.cost_and_grad(activations, gradients, costs,
      flat_labels, label_lengths,
      input_lengths);
  else
    return ctc.score_forward(activations, costs, flat_labels,
      label_lengths, input_lengths);

}


ctcStatus_t FUN(get_workspace_size)(const int* const label_lengths,
  const int* const input_lengths,
  int alphabet_size, int minibatch,
  ctcOptions options,
  size_t* size_bytes)
{
  if (label_lengths == nullptr ||
    input_lengths == nullptr ||
    size_bytes == nullptr ||
    alphabet_size <= 0 ||
    minibatch <= 0)
    return CTC_STATUS_INVALID_VALUE;

  // This is the max of all S and T for all examples in the minibatch.
  int maxL = *std::max_element(label_lengths, label_lengths + minibatch);
  int maxT = *std::max_element(input_lengths, input_lengths + minibatch);

  const int S = 2 * maxL + 1;

  *size_bytes = 0;

  // GPU storage
  //nll_forward, nll_backward
  *size_bytes += 2 * sizeof(Dtype) * minibatch;

  //repeats
  *size_bytes += sizeof(int) * minibatch;

  //label offsets
  *size_bytes += sizeof(int) * minibatch;

  //utt_length
  *size_bytes += sizeof(int) * minibatch;

  //label lengths
  *size_bytes += sizeof(int) * minibatch;

  //labels without blanks - overallocate for now
  *size_bytes += sizeof(int) * maxL * minibatch;

  //labels with blanks
  *size_bytes += sizeof(int) * S * minibatch;

  //alphas
  *size_bytes += sizeof(Dtype) * S * maxT * minibatch;

  //denoms
  *size_bytes += sizeof(Dtype) * maxT * minibatch;

  //probs (since we will pass in activations)
  *size_bytes += sizeof(Dtype) * alphabet_size * maxT * minibatch;


  return CTC_STATUS_SUCCESS;
}

void FUN(ExtractInputData)(int T_, int N_, int C_, int blank_index_, const Dtype* seq_ind_data, const Dtype* labels_data, vector<int>* flat_labels, vector<int>* label_lengths, vector<int>* input_lengths) {
  const Dtype* seq_ind = seq_ind_data;
  const Dtype* target_seq = labels_data;
  flat_labels->clear();
  flat_labels->reserve(T_ * N_);  // maximum required
  label_lengths->resize(N_);
  input_lengths->resize(N_);
  // compute the sequence length and label length
  int* seq_len = input_lengths->data();
  int* label_len = label_lengths->data();
  int label_offset = 0;
  //if (blank_index_ == -1) {
  if (blank_index_ == 0) {//modified by jxs
    label_offset = 1;
  }
  for (int n = 0; n < N_; ++n) {
    seq_len[n] = T_;  // default value is maximal allowed length
    label_len[n] = T_;  // default value is maximal allowed length
    const Dtype* seq = seq_ind + n;
    const Dtype* label = target_seq + n;
    // sequence indicators start with seq == 0.0 to indicate the start of a
    // sequence. Skip at t = 0, so start at t = 1
    seq += N_;
    for (int t = 1; t < T_; ++t) {
      if (static_cast<int>(*seq + 0.5) == 0) {
        seq_len[n] = t;
        break;
      }
      seq += N_;
    }
    // label indicators are negative if the sequence has ended
    for (int t = 0; t < T_; ++t) {
      if (*label < 0.0) {
        label_len[n] = t;
        break;
      }
      // Note that the blank label will be 0
      flat_labels->push_back(static_cast<int>(*label + 0.5) + label_offset);
      label += N_;
    }
    // if the label length is 0, the seq_len is 1 (0 following 0)
    // set seq_len to 0 in this case aswell, to skip this example
    if (label_len[n] == 0) {
      CHECK_LE(seq_len[n], 1);
      seq_len[n] = 0;
    }
    CHECK_LE(label_len[n], seq_len[n])
      << "The label length must be smaller or equals the sequence length!";
  }
}

void FUN(warp_ctc_loss_fwd)(int T_, int N_, int C_, int count, int blank_index_,
  const Dtype* bottom0_data, Dtype* bottom0_mdiff, const Dtype* bottom1_data,
  const Dtype* bottom2_data, const Dtype* bottom3_data, Dtype* top) {
  const Dtype* activations = bottom0_data;
  Dtype* gradients = bottom0_mdiff;
  const int alphabet_size = C_;
  const int minibatch = N_;
  int bottom_size = (bottom0_data != NULL) + (bottom1_data != NULL) + (bottom2_data != NULL) + (bottom3_data != NULL);
  vector<int> flat_labels_;
  vector<int> label_lengths_;
  vector<int> input_lengths_;
  label_lengths_.resize(N_);
  input_lengths_.resize(N_);
  vector<Dtype> costs(N_);
  flat_labels_.clear();
  if (bottom_size == 2) {//bottom[0]=activations, bottom[1] is labels, shape: Batchsize*seq len
    const Dtype* label_seq_d = bottom1_data;
    int label_len_per_batch = count / N_;
    for (int n = 0; n < N_; ++n) {
      int curlen = 0;
      for (int l = 0; l < label_len_per_batch; ++l) {
        int label = (int)label_seq_d[n * label_len_per_batch + l];
        if (label <= blank_index_) {
          continue;
        }
        flat_labels_.push_back(label);
        curlen++;
      }
      label_lengths_[n] = curlen;
      input_lengths_[n] = T_;
    }
  }
  else if (bottom_size == 3) {
    FUN(ExtractInputData)(T_, N_, C_, blank_index_, bottom1_data, bottom2_data, &flat_labels_, &label_lengths_, &input_lengths_);
  }
  else if (bottom_size == 4) {
    //Blob* seq_len_blob = bottom1_data;
    //Blob* lab_len_blob = bottom2_data;
    //Blob* label_seq_blob = bottom3_data;
    const Dtype* seq_len_d = bottom1_data;
    const Dtype* lab_len_d = bottom2_data;
    const Dtype* label_seq_d = bottom3_data;
    int accumulated = 0;
    int label_len_per_batch = count / N_;
    //CHECK_EQ(seq_len_blob->count(), lab_len_blob->count());
    for (int i = 0; i < count; ++i) {
      label_lengths_[i] = (int)lab_len_d[i];
      input_lengths_[i] = (int)seq_len_d[i];
      accumulated += (int)lab_len_d[i];
    }
    flat_labels_.clear();
    flat_labels_.reserve(accumulated);
    for (int n = 0; n < N_; ++n) {
      for (int t = 0; t < label_lengths_[n]; ++t) {
        flat_labels_.push_back((int)label_seq_d[n*label_len_per_batch + t]);
      }
    }
  }
  else {
    LOG(FATAL) << "Unsupported blobs shape";
  }
  //remove repeat blank labels
  size_t workspace_alloc_bytes_;
  ctcOptions options;

  hipStream_t stream;
  CHECK_EQ(hipStreamCreate(&stream), hipSuccess);
  options.loc = CTC_GPU;
  options.stream = stream;

  options.blank_label = blank_index_;
  ctcStatus_t status = FUN(get_workspace_size)(label_lengths_.data(),
    input_lengths_.data(),
    alphabet_size,
    minibatch,
    options,
    &workspace_alloc_bytes_);
  CHECK_EQ(status, CTC_STATUS_SUCCESS) << "CTC Error: " << ctcGetStatusString(status);
  Buffer workspace_[1] = { 0 };
  if (workspace_->size< workspace_alloc_bytes_) {
    gpu_ReAlloc(workspace_, workspace_alloc_bytes_ * sizeof(char));
  }

  //cuda_compute_ctc_loss;
  status = FUN(compute_ctc_loss)(activations,
    gradients,
    flat_labels_.data(),
    label_lengths_.data(),
    input_lengths_.data(),
    alphabet_size,
    minibatch,
    costs.data(),
    workspace_->data,
    options
  );
  CHECK_EQ(status, CTC_STATUS_SUCCESS) << "CTC Error: " << ctcGetStatusString(status);
  // output loss
  Dtype loss;// = top_mdata()[0];
  loss = 0;
  int num = 1;
  for (int n = 0; n < N_; ++n) {
    if (costs[n] < std::numeric_limits<Dtype>::infinity()) {
      loss += costs[n];
      ++num;
    }
  }
  if (num==1) {
    int asdf = 0;
  }
  loss /= num;
  if (isnan(loss)) {
    int asdf = 0;
  }
  *top = loss;
  Free(workspace_);
#if 0
  int gcnt = bottom[0]->count();
  Dtype sumg = 0;
  for (int i = 0; i < gcnt; i++) {
    sumg += fabs(gradients[i]);
  }
  //LOG(INFO) << "mean ctc loss=" << loss << ",N_="<<N_<<",num="<<num << ", mean gradients="<<sumg/gcnt;
#endif
  CHECK_EQ(hipStreamDestroy(stream), hipSuccess);
  return;
}
